#include "hip/hip_runtime.h"
// cuda_c_bench.cu
// nvcc -arch=sm_80 -O3 cuda_c_bench.cu
#include <stdio.h>
#include "sys/time.h"

#define GPU_ID 7

// #define USE_SINGLE_PRECISION    /* Comment this line using "!" if you want to use double precision.  */
#ifdef USE_SINGLE_PRECISION
#define DAT     float
#define PRECIS  4
#define SC      2
#define my_pow(A,b) powf(A,b)
#else
#define DAT     double
#define PRECIS  8
#define SC      1
#define my_pow(A,b) pow(A,b)
#endif
#define zeros(A,nx,ny)  DAT *A##_d,*A##_h; A##_h = (DAT*)malloc((nx)*(ny)*sizeof(DAT)); \
                        for(i=0; i < (nx)*(ny); i++){ A##_h[i]=(DAT)0.0; }              \
                        hipMalloc(&A##_d      ,(nx)*(ny)*sizeof(DAT));                 \
                        hipMemcpy( A##_d,A##_h,(nx)*(ny)*sizeof(DAT),hipMemcpyHostToDevice);
#define  free_all(A)    free(A##_h);hipFree(A##_d);

#define BLOCK_X   32
#define BLOCK_Y   8

#define FACT      32
#define NX       (SC*FACT*1024)
#define NY       (   FACT*1024)

unsigned int GRID_X = 1 + ((NX - 1) / BLOCK_X);
unsigned int GRID_Y = 1 + ((NY - 1) / BLOCK_Y);

const size_t nx = NX;
const size_t ny = NY;
const int nt = 100;

// Timer
double timer_start = 0;
double cpu_sec(){ struct timeval tp; gettimeofday(&tp,NULL); return tp.tv_sec+1e-6*tp.tv_usec; }
void   tic(){ timer_start = cpu_sec(); }
double toc(){ return cpu_sec()-timer_start; }
void   tim(const char *what, double n){ double s=toc(); printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n"); }

// void timPrint(const char *what, double n, int nx, int ny){
//   double s=toc();
//   printf("%s: %8.3f seconds",what,s);if(n>0)printf(", %8.3f GB/s", n/s); printf("\n");
//   FILE*fid; fid=fopen("PERF_memcpy.dat","a"); fprintf(fid,"nx=%d ny=%d GBs=%1.4f  time_s=%1.4f \n", nx, ny, n/s, s); fclose(fid);
// }

void  clean_cuda(){ 
    hipError_t ce = hipGetLastError();
    if(ce != hipSuccess){ printf("ERROR launching GPU C-CUDA program: %s\n", hipGetErrorString(ce)); hipDeviceReset();}
}

__global__ void memcopy(DAT*A, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    if (iy<ny && ix<nx) A[ix + iy*nx] = A[ix + iy*nx] + (DAT)1.0;
}

__global__ void memcopy_triad(DAT*A, DAT*B, DAT*C, DAT s, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    if (iy<ny && ix<nx) A[ix + iy*nx] = B[ix + iy*nx] + (DAT)s*C[ix + iy*nx];
}

__global__ void memcopy_triad_pow_int(DAT*A, DAT*B, DAT*C, DAT s, const int pow_int, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    if (iy<ny && ix<nx) A[ix + iy*nx] = B[ix + iy*nx] + (DAT)s*my_pow(C[ix + iy*nx], pow_int);
}

__global__ void memcopy_triad_pow_float(DAT*A, DAT*B, DAT*C, DAT s, DAT pow_float, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    if (iy<ny && ix<nx) A[ix + iy*nx] = B[ix + iy*nx] + (DAT)s*my_pow(C[ix + iy*nx], pow_float);
}

__global__ void diff2D_step(DAT*A, DAT*B, DAT*C, DAT s, DAT dt, DAT _dx, DAT _dy, const int nx, const int ny){
    int ix = blockIdx.x*blockDim.x + threadIdx.x;
    int iy = blockIdx.y*blockDim.y + threadIdx.y;
    if (iy>0 && iy<(ny-1) && ix>0 && ix<(nx-1)){
        A[ix + iy*nx] = B[ix + iy*nx] + dt*(C[ix + iy*nx]*(
                 - ((-s*(B[ix+1 + iy*nx] - B[ix + iy*nx])*_dx) - (-s*(B[ix + iy*nx] - B[ix-1 + iy*nx])*_dx))*_dx
                 - ((-s*(B[ix + (iy+1)*nx] - B[ix + iy*nx])*_dy) - (-s*(B[ix + iy*nx] - B[ix + (iy-1)*nx])*_dy))*_dy));
    }
}

////////// main //////////
int main(){
    size_t i, it, N=nx*ny, mem=N*sizeof(DAT);
    time_t t;
    srand((unsigned) time(&t));
    dim3 grid, block;
    block.x = BLOCK_X; block.y = BLOCK_Y;
    grid.x  = GRID_X;  grid.y  = GRID_Y;
    int gpu_id=-1; gpu_id=GPU_ID; hipSetDevice(gpu_id); hipGetDevice(&gpu_id);
    hipDeviceReset(); hipDeviceSetCacheConfig(hipFuncCachePreferL1);  // set L1 to prefered
    printf("Process uses GPU with id %d.\n",gpu_id);
    printf("%lux%lu, %1.3f GB, %d iterations.\n", nx,ny, 1*mem/1024./1024./1024., nt);
    printf("launching (%ux%u) grid of (%ux%u) blocks.\n", grid.x, grid.y, block.x, block.y);
    // initializations  
    zeros(A, nx,ny);
    zeros(B, nx,ny);
    zeros(C, nx,ny);
    DAT s    = rand();
    int pow_int   = 4;
    DAT pow_float = 4.75;
    DAT _dx  = 1.0;
    DAT _dy  = 1.0;
    DAT  dt  = 1.0/10.0/4.1;
    // tests
    for(it=0; it<nt; it++){ 
        if (it==10){ tic(); }
        memcopy<<<grid, block>>>(A_d, nx, ny); hipDeviceSynchronize();
    }
    tim("Performance memcpy", mem*(nt-10)*2/1024./1024./1024.);

    for(it=0; it<nt; it++){ 
        if (it==10){ tic(); }
        memcopy_triad<<<grid, block>>>(A_d, B_d, C_d, s, nx, ny); hipDeviceSynchronize();
    }
    tim("Performance triad2D", mem*(nt-10)*3/1024./1024./1024.);

    for(it=0; it<nt; it++){ 
        if (it==10){ tic(); }
        memcopy_triad_pow_int<<<grid, block>>>(A_d, B_d, C_d, s, pow_int, nx, ny); hipDeviceSynchronize();
    }
    tim("Performance triad2D_pow_int", mem*(nt-10)*3/1024./1024./1024.);

    for(it=0; it<nt; it++){ 
        if (it==10){ tic(); }
        memcopy_triad_pow_float<<<grid, block>>>(A_d, B_d, C_d, s, pow_float, nx, ny); hipDeviceSynchronize();
    }
    tim("Performance triad2D_pow_float", mem*(nt-10)*3/1024./1024./1024.);

    for(it=0; it<nt; it++){ 
        if (it==10){ tic(); }
        diff2D_step<<<grid, block>>>(A_d, B_d, C_d, s, dt, _dx, _dy, nx, ny); hipDeviceSynchronize();
    }
    tim("Performance diff2D_step", mem*(nt-10)*3/1024./1024./1024.);

    free_all(A);
    free_all(B);
    free_all(C);
    clean_cuda();
    return 0;
}
